
#include <hip/hip_runtime.h>
/**
 * @file matrix_mult_naive_single.cu
 *
 * CUDA code to calculate D = A*B using a naive algorithm (single precision)
 *
 */


/** Main entry point.
 * Implements naive single precision matrix multiplication
 */
__global__ void matrix_mult_naive_single(
                      float * D,
                      const float * A,
                      const float * B,
                      const unsigned int A_B_sum_length,
                      const unsigned int columns_D,
                      const unsigned int rows_D) {

    // Work out which thread we are
    int row_D_idx = blockIdx.y * blockDim.y + threadIdx.y;
    int column_D_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform matrix multiplication
    if (row_D_idx < rows_D && column_D_idx < columns_D){
        float tmp_sum = 0.0;
        for (int k_idx = 0; k_idx < A_B_sum_length; k_idx++){
            tmp_sum += A[row_D_idx * A_B_sum_length + k_idx] * B[k_idx * columns_D + column_D_idx];
        }
        // Place in the output array
        D[row_D_idx * columns_D + column_D_idx] = tmp_sum;
    }
}

